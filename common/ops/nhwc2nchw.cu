#include "hip/hip_runtime.h"
#include "nhwc2nchw.h"

__global__ void transpose_kernel(
        const uint8_t* input,
        float* output,
        const int n,
        const int h,
        const int w,
        const float mean_0,
        const float mean_1,
        const float mean_2,
        const float var_0,
        const float var_1,
        const float var_2,
        const bool bgr_mode) {
    int stride = h * w;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int pn = idx / stride;
    if (pn >= n)
        return;

    const uint8_t* ip = input + idx * 3;
    float* op = output + idx + pn * 2 * stride;

    if (bgr_mode) {
        op[0] = (float)(ip[0] - mean_0) / var_0;
        op[stride] = (float)(ip[1] - mean_1) / var_1;
        op[2 * stride] = (float)(ip[2] - mean_2) / var_2;
    } else {
        op[0] = (float)(ip[2] - mean_0) / var_0;
        op[stride] = (float)(ip[1] - mean_1) / var_1;
        op[2 * stride] = (float)(ip[0] - mean_2) / var_2;
    }

}

void NHWC2NCHW(
        const uint8_t* input,
        float* output,
        const int n,
        const int h,
        const int w,
        const float mean_0,
        const float mean_1,
        const float mean_2,
        const float var_0,
        const float var_1,
        const float var_2,
        const bool bgr_mode) {
    transpose_kernel<<<(n * h * w - 1) / BLOCK + 1, BLOCK>>>(input, output, n, h, w,
                                                             mean_0, mean_1, mean_2,
                                                             var_0, var_1, var_2, bgr_mode);
}
